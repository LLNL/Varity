
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4,float var_5,float var_6) {
comp += ldexpf((+1.0284E34f / sqrtf((var_2 / cosf((-1.0213E25f * (var_3 + +1.8102E-43f)))))), 2);
for (int i=0; i < var_1; ++i) {
  float tmp_1 = +0.0f;
float tmp_2 = -1.1541E-42f;
float tmp_3 = (var_4 * (var_5 - -1.1607E-35f));
comp = tmp_3 / tmp_2 + tmp_1 / var_6 - -0.0f;
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7);
  hipDeviceSynchronize();

  return 0;
}
