
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, int var_1) {
if (comp == coshf(+1.4691E-36f)) {
  for (int i=0; i < var_1; ++i) {
    comp += -1.2061E10f / +1.2134E-37f / -1.9037E36f + -1.1056E35f;
}
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);

  compute<<<1,1>>>(tmp_1,tmp_2);
  hipDeviceSynchronize();

  return 0;
}
