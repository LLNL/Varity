
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, float var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12) {
if (comp <= (var_1 + cosf(+1.3081E28f / var_2 / (var_3 / var_4)))) {
  if (comp > (-0.0f * -0.0f)) {
    float tmp_1 = (var_5 + var_6 - (var_7 + (-1.8676E35f / (-1.3184E-37f / -1.0293E35f))));
comp = tmp_1 + (var_8 / (var_9 * var_10));
comp += sinhf((var_11 * -1.3226E35f));
comp += -1.7276E-44f * (var_12 / cosf(-0.0f * (-1.0656E-7f * -1.3901E36f / -1.6465E-42f)));
}
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13);
  hipDeviceSynchronize();

  return 0;
}
