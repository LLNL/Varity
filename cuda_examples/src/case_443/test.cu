
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, int var_1,float var_2,float* var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20,float var_21,float var_22,float var_23,float var_24,float var_25) {
if (comp == (var_2 / +1.8953E-43f)) {
  for (int i=0; i < var_1; ++i) {
    var_3[i] = asinf(-0.0f);
comp += var_3[i] + atan2f(powf(+1.6429E-44f / fabsf((-1.7514E5f * -1.4085E-35f)), fmodf((var_4 - var_5 - coshf(var_6 - -1.3860E-35f + var_7)), -1.9893E-43f * (+1.7807E-37f / sqrtf(+1.3247E-37f)))), (var_8 / +0.0f * (+1.7087E-37f / var_9)));
if (comp <= fmodf((var_10 + (+0.0f / var_11)), (-1.4604E-43f / var_12 * var_13))) {
  comp = var_14 / +1.9332E-43f;
}
if (comp <= (+1.6425E-41f * (var_15 - var_16))) {
  float tmp_1 = (-1.5774E7f - (var_17 - powf(-1.2596E13f / -1.8114E-12f, -1.9215E-37f * var_18)));
float tmp_2 = (-1.8062E-21f + (-1.6115E-41f * coshf((var_19 * var_20 + (var_21 + cosf(fmodf(+1.2369E-35f, expf(var_22 / var_23 - var_24))))))));
comp = tmp_2 * tmp_1 / (var_25 / +1.0353E24f * +1.6672E-42f);
}
}
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float* tmp_4 = initPointer( atof(argv[4]) );
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);
  float tmp_23 = atof(argv[23]);
  float tmp_24 = atof(argv[24]);
  float tmp_25 = atof(argv[25]);
  float tmp_26 = atof(argv[26]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23,tmp_24,tmp_25,tmp_26);
  hipDeviceSynchronize();

  return 0;
}
