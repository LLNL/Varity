
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, float var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15) {
if (comp <= var_1 - (var_2 + asinf(ceilf((var_3 / var_4))))) {
  comp += (+1.0969E-25f * asinf(var_5 / var_6));
float tmp_1 = asinf((var_7 + var_8 / (-1.6537E-16f * (-1.7489E-43f - -1.1432E-35f))));
comp += tmp_1 / var_9 * (-1.5681E-42f - (var_10 / +1.9191E35f - (-0.0f / +1.2688E-36f)));
if (comp < -1.1507E-35f / cosf((var_11 / (-0.0f + (var_12 * (-1.7008E-17f * +1.6368E-37f / var_13)))))) {
  comp += (var_14 - (-1.2497E-44f - (-1.9869E-27f / var_15 / -1.7224E-43f)));
}
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16);
  hipDeviceSynchronize();

  return 0;
}
