
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, int var_1,int var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float* var_16,float var_17,float var_18,float var_19,float var_20,float var_21) {
for (int i=0; i < var_1; ++i) {
  float tmp_1 = +1.7238E-13f;
comp += tmp_1 / (var_3 + (var_4 * coshf((+0.0f * (var_5 + (var_6 - (var_7 / logf(atan2f(+1.5160E-13f * var_8 - (var_9 * -1.2032E19f), -1.0343E-36f)))))))));
if (comp == logf((var_10 / var_11 + var_12 + (var_13 * -1.2225E36f)))) {
  comp += -0.0f * (var_14 - var_15 + +1.7355E35f);
}
for (int i=0; i < var_2; ++i) {
  comp += (-1.2430E-43f - var_17 * +1.7458E-37f - +1.2869E-36f);
var_16[i] = +1.2003E-29f;
comp = var_16[i] / var_18 * (var_19 - +1.9828E1f);
comp += var_20 - (var_21 + +1.6830E10f - (+1.1497E-37f + -1.0421E-44f));
}
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float* tmp_17 = initPointer( atof(argv[17]) );
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22);
  hipDeviceSynchronize();

  return 0;
}
