
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, float var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8) {
float tmp_1 = -1.0034E34f;
float tmp_2 = +0.0f;
comp += tmp_2 - tmp_1 * var_1 * sqrtf(var_2 * -1.2132E-37f);
comp = sqrtf(var_3 / atanf(+0.0f));
if (comp > (-0.0f - var_4 - -1.1122E-41f + +0.0f / (var_5 / var_6))) {
  comp = tanhf(+1.2248E-35f * +1.9634E-23f + (+1.7002E-10f / (var_7 + log10f((-1.0865E36f + (var_8 / +1.0608E-41f))))));
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9);
  hipDeviceSynchronize();

  return 0;
}
