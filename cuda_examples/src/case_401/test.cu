
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, float var_1,float var_2,float var_3,float var_4,float var_5) {
if (comp == var_1 - (var_2 * -1.9905E-29f / acosf(fabsf(-0.0f)))) {
  float tmp_1 = +1.4755E34f;
float tmp_2 = -1.0503E-42f;
comp += tmp_2 - tmp_1 / +1.1211E-37f + (+1.4038E-36f - -1.6116E35f);
if (comp <= +1.9457E-43f + -1.6375E-35f + var_3 / -1.9240E-41f / +1.6891E-37f) {
  comp = atanf(+1.8338E-35f);
comp += var_4 * var_5;
}
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6);
  hipDeviceSynchronize();

  return 0;
}
