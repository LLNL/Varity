
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4,float var_5,float var_6) {
if (comp <= var_2 + (-1.8131E36f + var_3)) {
  if (comp > atan2f((+1.0355E4f / (+1.3487E25f * atanf((var_4 / asinf(ldexpf(-1.0584E0f, 2)))))), (var_5 * (+1.9850E-26f - var_6 * -1.6918E-36f)))) {
    for (int i=0; i < var_1; ++i) {
      comp += -1.2283E-37f / +1.0912E-42f * -1.4820E-29f;
comp += +1.7001E-36f + (-0.0f + powf((+1.2842E-37f + asinf(+1.1682E-43f)), -1.4971E10f / (-0.0f - -1.1527E-43f)));
}
}
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7);
  hipDeviceSynchronize();

  return 0;
}
