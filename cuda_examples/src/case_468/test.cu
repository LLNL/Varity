
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13) {
if (comp >= var_2 / var_3 * -1.9642E34f) {
  if (comp >= (+0.0f / -1.5998E-43f + +1.6862E-27f + (var_4 / var_5))) {
    for (int i=0; i < var_1; ++i) {
      float tmp_1 = sinhf(-1.6157E-35f);
float tmp_2 = (var_6 * (-1.8705E-42f / var_7 - +0.0f));
comp += tmp_2 * tmp_1 / -1.2006E-41f * +1.5022E-36f * +1.3346E-43f - +1.9799E-37f * -1.3239E22f + -1.3287E36f;
comp += asinf(+1.1500E-37f);
if (comp < +1.7669E21f + ldexpf((var_8 * (-0.0f / +1.5812E-41f / (-1.3024E-44f - var_9))), 2)) {
  comp = -1.9185E-13f / var_10 * +1.9758E-43f * (var_11 / (var_12 * var_13));
}
}
}
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14);
  hipDeviceSynchronize();

  return 0;
}
