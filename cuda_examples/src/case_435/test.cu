
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, float var_1,float var_2,float var_3,int var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13) {
if (comp == sinf(+1.1894E-36f - powf(-1.3596E-44f + (var_1 + (+1.4869E-41f / var_2 + (var_3 * +0.0f))), +1.1176E-42f))) {
  float tmp_1 = +0.0f;
float tmp_2 = +1.8021E36f;
comp += tmp_2 / tmp_1 + var_5 + var_6 - +1.7849E-37f * +1.8890E-37f + -0.0f;
for (int i=0; i < var_4; ++i) {
  comp += +1.4990E7f / +1.1159E-37f;
comp = (-1.3709E5f + sinf((var_7 / -1.3219E-42f / var_8)));
comp = fabsf((var_9 - +1.2732E34f));
}
if (comp < -1.1817E-43f + (+1.1781E-1f / +1.0107E-44f)) {
  comp = asinf(var_10 + var_11 + (var_12 - +1.4480E-19f - var_13));
}
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  int tmp_5 = atoi(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14);
  hipDeviceSynchronize();

  return 0;
}
