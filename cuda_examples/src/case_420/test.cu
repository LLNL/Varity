
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, int var_1,int var_2,int var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float* var_13,float var_14,float var_15,float var_16,float* var_17,float var_18,float var_19,float var_20,float var_21,float var_22,float var_23,float var_24,float var_25,float var_26,float var_27,float* var_28,float var_29,float var_30) {
if (comp == (var_4 + -1.8522E-37f + var_5 - (var_6 / cosf((-1.2784E17f * (var_7 / tanhf(var_8 + -0.0f - var_9))))))) {
  comp += (-1.1977E35f * var_10 / (+1.9911E-35f * (+1.2078E-36f * atanf((var_11 / +1.4141E-41f + var_12)))));
for (int i=0; i < var_1; ++i) {
  var_13[i] = (var_14 / log10f(+1.0240E-35f));
comp += var_13[i] - var_15 + -0.0f;
comp += +1.4381E-41f / +1.2882E-36f * var_16 / -1.5557E-15f;
}
for (int i=0; i < var_2; ++i) {
  var_17[i] = +0.0f;
comp = var_17[i] / (-1.5308E-36f - (-0.0f + var_18 + (+1.4126E-37f * var_19)));
float tmp_1 = var_20 * var_21 * +0.0f * (-0.0f / +1.7610E-37f / var_22);
comp = tmp_1 * var_23 / var_24 * (var_25 - (var_26 - var_27));
}
for (int i=0; i < var_3; ++i) {
  var_28[i] = +0.0f;
comp += var_28[i] - +1.6726E34f - var_29 * (+1.3177E-37f + (-1.9295E36f / var_30));
}
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  int tmp_4 = atoi(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float* tmp_14 = initPointer( atof(argv[14]) );
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float* tmp_18 = initPointer( atof(argv[18]) );
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);
  float tmp_23 = atof(argv[23]);
  float tmp_24 = atof(argv[24]);
  float tmp_25 = atof(argv[25]);
  float tmp_26 = atof(argv[26]);
  float tmp_27 = atof(argv[27]);
  float tmp_28 = atof(argv[28]);
  float* tmp_29 = initPointer( atof(argv[29]) );
  float tmp_30 = atof(argv[30]);
  float tmp_31 = atof(argv[31]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23,tmp_24,tmp_25,tmp_26,tmp_27,tmp_28,tmp_29,tmp_30,tmp_31);
  hipDeviceSynchronize();

  return 0;
}
