
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, int var_1,int var_2,int var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float* var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20,float var_21,float var_22) {
for (int i=0; i < var_1; ++i) {
  if (comp < coshf(+1.8494E36f / (var_4 / -1.7419E35f))) {
    if (comp >= var_5 / (-1.2732E34f * (-1.6059E-37f / (+1.0834E-44f * var_6)))) {
      comp += (var_7 / var_8 - +1.8188E35f);
comp += var_9 + (-1.2492E-35f / var_10);
float tmp_1 = ceilf(-1.6458E36f);
comp = tmp_1 / var_11 * asinf(-1.8096E36f);
for (int i=0; i < var_2; ++i) {
  var_12[i] = +1.8832E-42f;
comp = var_12[i] + (+1.6264E20f - var_13 - (var_14 * +1.1822E-35f));
comp += var_15 + +1.3064E-28f;
}
for (int i=0; i < var_3; ++i) {
  comp = (+1.5325E35f + (var_16 - ldexpf(cosf(+0.0f), 2)));
comp = sinhf((+1.6014E-43f - acosf(var_17 * (var_18 - (+1.7912E-43f * -1.9747E-36f / var_19)))));
comp = ceilf(-1.3857E-35f * (-0.0f * var_20));
comp = (var_21 * var_22);
}
if (comp < +1.0727E-37f / (-1.8503E-35f * fmodf((+1.0860E36f - -1.9976E-35f), -1.0564E9f))) {
  comp += atan2f(-1.3276E-41f, +1.4777E-44f);
comp = atanf(-0.0f - -1.8053E-42f - +0.0f / +1.1653E36f);
}
}
}
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  int tmp_4 = atoi(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float* tmp_13 = initPointer( atof(argv[13]) );
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);
  float tmp_23 = atof(argv[23]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23);
  hipDeviceSynchronize();

  return 0;
}
