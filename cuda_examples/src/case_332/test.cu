
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, float var_1,int var_2,int var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10) {
if (comp > +0.0f - var_1) {
  comp += -1.5893E34f * (var_4 * +1.4138E34f);
for (int i=0; i < var_2; ++i) {
  comp = var_5 + -1.6156E36f / var_6;
}
for (int i=0; i < var_3; ++i) {
  comp = -0.0f - fmodf((-1.7388E-42f * (var_7 / var_8 / (+1.8761E-17f - +1.1425E-35f))), var_9 / +0.0f / (-1.2067E35f - (-1.0432E-36f * var_10)));
}
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  int tmp_3 = atoi(argv[3]);
  int tmp_4 = atoi(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11);
  hipDeviceSynchronize();

  return 0;
}
