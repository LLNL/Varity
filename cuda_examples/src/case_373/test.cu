
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, float var_1,float var_2,int var_3,int var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float* var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float* var_20,float var_21) {
if (comp == (-0.0f / var_1 / expf(acosf(-1.2513E-43f - var_2 / +1.9779E-37f)))) {
  comp = (+1.8974E-44f + +1.2493E-37f + -1.0347E-30f * (-1.4569E27f - var_5));
comp = +1.1623E-5f * log10f(var_6 + (var_7 + -1.3131E-37f * +1.7783E-37f));
comp += var_8 - (-0.0f + fabsf(-1.0131E21f + atanf(powf(coshf((var_9 - (-1.4074E-44f - var_10 / -1.6276E34f - +1.0109E-7f))), (var_11 - log10f(+1.7576E-35f))))));
for (int i=0; i < var_3; ++i) {
  var_12[i] = (var_13 / (var_14 * +1.4414E-42f / sinhf(-1.5031E-44f - (-0.0f - var_15 - (var_16 / (-1.4357E35f - -1.7691E36f))))));
comp += var_12[i] / -1.0193E-35f * var_17;
comp += +0.0f - -0.0f / (var_18 * -1.2206E36f);
comp += var_19 * -1.5791E-43f;
}
for (int i=0; i < var_4; ++i) {
  var_20[i] = +1.0155E35f;
comp = var_20[i] / -0.0f + -1.0106E35f * -1.8495E36f * var_21;
comp += +1.3088E34f * +1.2880E-23f - -1.1207E10f;
}
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  float tmp_3 = atof(argv[3]);
  int tmp_4 = atoi(argv[4]);
  int tmp_5 = atoi(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float* tmp_13 = initPointer( atof(argv[13]) );
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float* tmp_21 = initPointer( atof(argv[21]) );
  float tmp_22 = atof(argv[22]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22);
  hipDeviceSynchronize();

  return 0;
}
