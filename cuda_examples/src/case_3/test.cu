
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4,float var_5,float var_6) {
for (int i=0; i < var_1; ++i) {
  if (comp == var_2 - -1.9330E-35f / +1.4582E34f - (-0.0f + -1.3565E-36f)) {
    comp = asinf((-1.4529E-9f / +1.7293E-43f + -1.4389E-41f));
comp = var_3 / +1.1878E-41f / (+1.3303E-43f - -1.7053E-44f);
comp = coshf((-1.4761E-41f + var_4 / var_5));
comp += var_6 * (+1.4955E35f / -0.0f - (+1.5835E-41f * +1.2444E-43f));
}
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7);
  hipDeviceSynchronize();

  return 0;
}
