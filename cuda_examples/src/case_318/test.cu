
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, float var_1,float var_2,int var_3,int var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17) {
if (comp <= cosf(-1.3183E-43f * var_1)) {
  if (comp == (+1.4693E-42f / -1.3452E-37f + ldexpf(expf(var_2 / -1.3044E-6f), 2))) {
    float tmp_1 = +0.0f * (var_5 * +1.3045E-41f - -1.6079E-1f + +1.5984E-44f);
comp = tmp_1 + atan2f(+1.3203E-42f + var_6, -1.1528E36f);
if (comp <= +1.5156E36f * expf(var_7 * -1.2909E35f)) {
  comp = -1.7182E-37f - (var_8 - (+1.5009E-35f - +1.2950E35f - -1.6396E-21f));
float tmp_2 = -0.0f;
comp = tmp_2 - var_9 * +1.4362E35f;
}
for (int i=0; i < var_3; ++i) {
  comp = var_10 * -1.5497E34f - sinhf(sqrtf(expf(-0.0f - (var_11 * var_12))));
comp += (var_13 + sqrtf((+1.9559E35f + var_14)));
}
for (int i=0; i < var_4; ++i) {
  float tmp_3 = var_15 + tanhf(logf(-1.5079E16f));
float tmp_4 = +1.4164E-36f;
comp = tmp_4 + tmp_3 / +1.2844E7f * var_16 - +1.2031E-41f;
comp = (var_17 * +1.0304E-44f);
}
}
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  float tmp_3 = atof(argv[3]);
  int tmp_4 = atoi(argv[4]);
  int tmp_5 = atoi(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18);
  hipDeviceSynchronize();

  return 0;
}
