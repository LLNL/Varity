
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, float var_1,float var_2,float var_3,float var_4,float var_5,int var_6,float var_7,float var_8,float var_9,float var_10) {
if (comp >= (var_1 - (+1.3761E-43f / (+1.3829E-41f / -1.3286E-41f)))) {
  if (comp == ceilf(var_2 + (-1.8982E-35f + var_3 / (var_4 - var_5)))) {
    comp = (-0.0f / var_7 + powf((var_8 * cosf(acosf(+1.8122E-25f + (+1.2258E-37f * (var_9 - +1.6384E-36f))))), -0.0f));
for (int i=0; i < var_6; ++i) {
  comp += (var_10 / asinf(sinf(ldexpf(+1.4889E-42f, 2))));
}
}
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  int tmp_7 = atoi(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11);
  hipDeviceSynchronize();

  return 0;
}
