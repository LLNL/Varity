
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17) {
float tmp_1 = (var_2 * -1.3966E-43f / +1.4189E-42f - var_3);
comp += tmp_1 - logf(var_4 * (+1.9616E35f * (var_5 * (var_6 / (var_7 * var_8)))));
for (int i=0; i < var_1; ++i) {
  float tmp_2 = +1.7813E-42f;
comp = tmp_2 + var_9 * -1.9417E35f * +1.3835E-37f;
}
if (comp == (var_10 + var_11)) {
  comp = atanf(asinf((+1.8682E36f / (+1.8320E-26f / (var_12 + var_13)))));
comp += (-1.7855E-37f + var_14);
comp = var_15 * ldexpf((+1.7775E26f + var_16 + -1.5048E-21f + var_17), 2);
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18);
  hipDeviceSynchronize();

  return 0;
}
