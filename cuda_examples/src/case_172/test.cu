
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13) {
if (comp == (+1.9791E7f * coshf((var_2 + +0.0f)))) {
  comp += atan2f(floorf(+1.4721E-44f), -1.3111E-36f * ldexpf((+1.8350E-35f * var_3 - var_4), 2));
float tmp_1 = +1.0002E28f;
float tmp_2 = -1.2076E-35f;
comp += tmp_2 * tmp_1 / sinf(-0.0f);
if (comp == var_5 + -0.0f * asinf(+1.6377E-41f)) {
  float tmp_3 = -1.2597E10f;
comp = tmp_3 / powf(-1.2530E7f, -1.8301E-44f);
}
if (comp >= atan2f(atanf((var_6 / (var_7 - (var_8 - +1.9054E35f * +1.0821E-44f - var_9)))), +0.0f)) {
  comp = (var_10 - (-0.0f / cosf(+0.0f)));
}
for (int i=0; i < var_1; ++i) {
  float tmp_4 = -1.7750E-18f / -1.2021E-43f;
comp = tmp_4 * (+1.3609E-19f + +0.0f);
comp += var_11 - (+0.0f * +1.1393E4f);
comp = (var_12 - var_13);
}
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14);
  hipDeviceSynchronize();

  return 0;
}
