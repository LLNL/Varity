
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4,int var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17) {
if (comp == (-0.0f * (-1.4378E-36f + var_2 + (-0.0f - -1.1927E-36f)))) {
  for (int i=0; i < var_1; ++i) {
    if (comp <= (var_3 * var_4 / asinf(-0.0f))) {
      comp += fmodf(+1.0959E-6f - (-1.3753E-36f * sqrtf(var_6 / var_7)), (var_8 / (-1.8739E-41f * var_9 / expf(-1.0283E-36f / (var_10 / var_11)))));
for (int i=0; i < var_5; ++i) {
  comp += (var_12 * -1.4153E-37f);
}
if (comp > (var_13 / -1.0419E-37f)) {
  float tmp_1 = -1.4701E36f;
comp += tmp_1 * (-0.0f + (-1.1467E-41f * (var_14 / (var_15 + -0.0f))));
comp = +1.4184E36f + var_16 / var_17;
}
}
}
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  int tmp_6 = atoi(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18);
  hipDeviceSynchronize();

  return 0;
}
