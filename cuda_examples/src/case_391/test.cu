
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, int var_1,int var_2,int var_3,float var_4,float var_5,float var_6,float var_7,int var_8,float var_9,float* var_10,float var_11,float var_12,float var_13,float var_14) {
if (comp > (var_4 - -1.6884E-11f - powf(-0.0f + (var_5 - sinhf(-1.9661E34f / (-1.4342E-36f / (var_6 * -1.5904E-35f / (var_7 + +0.0f))))), -1.6178E-35f))) {
  for (int i=0; i < var_1; ++i) {
    for (int i=0; i < var_2; ++i) {
      for (int i=0; i < var_3; ++i) {
        comp += (+1.1944E-16f - fmodf((+1.3415E-35f * (+1.6316E-42f - +1.1709E36f)), -1.0730E36f - var_9));
if (comp == coshf((-0.0f / +1.0872E36f))) {
  comp = sinhf(-1.4058E-43f);
}
for (int i=0; i < var_8; ++i) {
  var_10[i] = +1.8297E36f;
comp = var_10[i] - var_11 * expf((var_12 / var_13 - var_14));
}
}
}
}
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  int tmp_4 = atoi(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  int tmp_9 = atoi(argv[9]);
  float tmp_10 = atof(argv[10]);
  float* tmp_11 = initPointer( atof(argv[11]) );
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15);
  hipDeviceSynchronize();

  return 0;
}
