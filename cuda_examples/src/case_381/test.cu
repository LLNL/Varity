
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, int var_1,int var_2,float var_3,float var_4,float var_5,float* var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20,float var_21,float var_22,float var_23,float var_24,float var_25,float var_26) {
for (int i=0; i < var_1; ++i) {
  float tmp_1 = +1.5495E-20f * -0.0f - +1.7373E34f + -1.2227E-41f;
comp += tmp_1 / powf(logf(+1.4807E-42f - (var_3 - var_4 + (-1.1452E-43f / -1.8998E5f))), (var_5 / -1.6529E34f));
for (int i=0; i < var_2; ++i) {
  var_6[i] = +1.6711E-42f;
comp += var_6[i] + (var_7 + (-0.0f + cosf(-1.3084E-36f / var_8 / +1.0464E-37f - var_9)));
comp += (var_10 * +1.1323E13f);
}
if (comp < (-1.7168E0f - ldexpf((var_11 / +1.2724E-43f * (+1.2128E-20f + (+1.5143E-36f + fabsf(-1.8362E-35f / var_12)))), 2))) {
  float tmp_2 = (var_13 * ceilf(var_14 + +1.0084E-35f));
comp += tmp_2 / floorf(var_15 + +1.9434E29f - var_16);
comp += (var_17 / (+1.1753E-44f - var_18 + +1.6236E-15f * atanf((var_19 / var_20 - (-1.9437E-44f / atanf(var_21 * +1.9941E36f + (var_22 * var_23 - var_24)))))));
comp += (-1.9196E-13f + var_25 - -1.8491E4f + var_26);
}
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float* tmp_7 = initPointer( atof(argv[7]) );
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);
  float tmp_23 = atof(argv[23]);
  float tmp_24 = atof(argv[24]);
  float tmp_25 = atof(argv[25]);
  float tmp_26 = atof(argv[26]);
  float tmp_27 = atof(argv[27]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23,tmp_24,tmp_25,tmp_26,tmp_27);
  hipDeviceSynchronize();

  return 0;
}
