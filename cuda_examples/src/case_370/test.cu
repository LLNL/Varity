
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, float var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20) {
if (comp > var_1 - var_2) {
  if (comp <= ldexpf((var_3 + +0.0f), 2)) {
    float tmp_1 = (var_4 - var_5);
float tmp_2 = +0.0f * var_6;
comp += tmp_2 * tmp_1 - floorf(+1.2953E-9f + (-0.0f + var_7 / (-1.9780E36f - powf(var_8 * +1.6747E-44f - var_9 / fmodf(-1.9290E34f / var_10 - -0.0f, (-1.0991E-44f - sinhf(var_11 * var_12 * -1.3232E-36f))), -1.6650E-35f * (+1.2151E-42f - (var_13 * (+1.8866E36f * (-1.7231E-41f * var_14))))))));
comp += +1.2812E10f / var_15;
if (comp <= atanf(-1.0044E-42f + (var_16 * -1.9840E8f * +1.6481E35f - var_17))) {
  float tmp_3 = +0.0f;
comp = tmp_3 / var_18 - var_19 - -1.3206E-36f - var_20 / -1.0774E-41f;
}
}
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21);
  hipDeviceSynchronize();

  return 0;
}
