
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, float var_1,float var_2,int var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15) {
if (comp > -1.6624E34f + (var_1 * var_2 * +1.5281E34f)) {
  float tmp_1 = +1.7184E-36f;
comp = tmp_1 + (var_4 - -1.5998E-41f);
comp += (var_5 * var_6);
if (comp <= atanf(+1.7906E-44f)) {
  comp += (-1.4259E34f - fmodf((-1.8190E-36f + (var_7 * var_8)), +1.7298E-43f));
comp += (-1.0381E-18f - var_9);
}
for (int i=0; i < var_3; ++i) {
  comp += (-0.0f + +1.3330E-7f);
comp += sqrtf((+1.9270E-43f + var_10 + (var_11 + -1.0902E35f / +1.3433E-37f)));
float tmp_2 = var_12 + var_13 * var_14;
comp += tmp_2 - (-1.3125E-37f * +0.0f - (-1.1000E-44f + (+1.7340E-41f + var_15)));
}
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  float tmp_3 = atof(argv[3]);
  int tmp_4 = atoi(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16);
  hipDeviceSynchronize();

  return 0;
}
