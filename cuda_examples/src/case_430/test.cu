
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float* var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20) {
comp = sqrtf((-1.1830E34f + var_2));
float tmp_1 = -1.6861E25f + -1.9280E-43f + +1.0388E34f / (var_3 / var_4);
comp = tmp_1 + var_5 + (-1.3749E15f + floorf(+1.2918E-44f));
comp += var_6 / (var_7 + +1.4018E-44f * var_8);
if (comp > -1.2561E-41f / -1.6609E35f / (var_9 / var_10)) {
  comp = sinf(-1.3345E-16f - -1.9315E-42f);
comp += -1.0176E-42f - +0.0f;
}
for (int i=0; i < var_1; ++i) {
  var_11[i] = +0.0f;
float tmp_2 = -1.4970E34f;
float tmp_3 = var_12 - (-1.4811E-37f - (-0.0f * sqrtf(var_13 + (-1.8149E-43f + (-1.4461E-42f - (+1.7745E34f - (var_14 - var_15)))))));
comp = tmp_3 - tmp_2 * var_11[i] * (var_16 + -1.5250E-35f + -1.3596E-44f - -0.0f);
}
if (comp > +1.9908E34f - expf(-1.6164E13f)) {
  comp = (var_17 + var_18 + fmodf(+1.2267E-35f * var_19, atanf((-1.6750E10f + +1.9142E-37f + -1.0010E-35f + -1.9315E-36f + (var_20 - -1.4851E-44f)))));
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float* tmp_12 = initPointer( atof(argv[12]) );
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21);
  hipDeviceSynchronize();

  return 0;
}
