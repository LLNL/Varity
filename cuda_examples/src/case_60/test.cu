
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, float var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9) {
comp = -1.9727E-37f - atan2f((+1.8066E19f - var_1 + var_2), -1.4946E-35f + (var_3 - sinhf(var_4 * (var_5 * var_6))));
comp = (-0.0f / (var_7 - -1.3422E-42f - -1.4089E-37f));
comp += -1.5327E36f / var_8 * var_9;
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10);
  hipDeviceSynchronize();

  return 0;
}
