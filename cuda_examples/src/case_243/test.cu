
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, int var_1,int var_2,float var_3,float var_4,float var_5,float var_6) {
for (int i=0; i < var_1; ++i) {
  if (comp == (-0.0f * var_3)) {
    if (comp == fabsf(+1.8912E36f)) {
      for (int i=0; i < var_2; ++i) {
        float tmp_1 = +1.0060E-36f;
comp += tmp_1 * ldexpf((var_4 - +1.4224E-30f - (var_5 / +1.6304E35f)), 2);
comp = (var_6 - -0.0f + cosf(powf(-1.3475E-43f + (+1.4059E36f - +1.4890E-35f), -1.2542E-44f)));
}
}
}
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7);
  hipDeviceSynchronize();

  return 0;
}
