
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, float var_1,int var_2,int var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float* var_19,float var_20,float var_21,float var_22,float var_23,float var_24) {
if (comp <= logf((var_1 - sinhf(+1.3758E22f)))) {
  comp = (var_4 - var_5);
comp += (var_6 * (-1.1832E35f * var_7 * var_8));
for (int i=0; i < var_2; ++i) {
  comp = (var_9 - (var_10 + acosf(var_11 * -1.6475E-43f)));
comp += asinf(+1.2860E-35f);
comp = fmodf(atan2f((var_12 - (var_13 + -0.0f - (var_14 * var_15 / -1.7790E14f))), var_16 + +1.6107E36f / -0.0f / +1.8488E36f * +1.4295E36f), -1.8084E-37f);
comp = -1.7217E35f / (+1.5718E-36f + (var_17 + (-1.0460E-41f - var_18)));
}
for (int i=0; i < var_3; ++i) {
  float tmp_1 = var_20 - -1.0851E34f;
var_19[i] = (-1.6743E-44f * +0.0f / (-1.3048E34f * +1.4836E-37f));
comp += var_19[i] - tmp_1 / (var_21 + var_22);
}
if (comp == fabsf((+1.0207E-36f / -1.1486E-41f))) {
  comp = -1.3839E-41f + (+1.5019E36f + (var_23 + -0.0f));
float tmp_2 = (-0.0f / (+1.9028E35f * +1.3077E-35f));
comp += tmp_2 + powf(+1.9011E-44f, (var_24 / +1.1185E35f));
}
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  int tmp_3 = atoi(argv[3]);
  int tmp_4 = atoi(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float* tmp_20 = initPointer( atof(argv[20]) );
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);
  float tmp_23 = atof(argv[23]);
  float tmp_24 = atof(argv[24]);
  float tmp_25 = atof(argv[25]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23,tmp_24,tmp_25);
  hipDeviceSynchronize();

  return 0;
}
