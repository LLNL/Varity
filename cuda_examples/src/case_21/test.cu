
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, int var_1,int var_2,float* var_3,int var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12) {
for (int i=0; i < var_1; ++i) {
  for (int i=0; i < var_2; ++i) {
    var_3[i] = var_5 / (-1.8028E35f + sinhf(+0.0f * +1.3585E4f - (var_6 * var_7)));
comp += var_3[i] + (var_8 - expf(coshf(+1.9783E-41f)));
comp += var_9 - -1.6863E-43f / +1.5580E-41f;
for (int i=0; i < var_4; ++i) {
  comp += var_10 + var_11;
comp = (-0.0f + var_12);
}
}
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  float* tmp_4 = initPointer( atof(argv[4]) );
  int tmp_5 = atoi(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13);
  hipDeviceSynchronize();

  return 0;
}
