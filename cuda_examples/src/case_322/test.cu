
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, int var_1,int var_2,int var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float* var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16) {
if (comp == logf(asinf(logf((var_4 / (-1.2967E34f - -1.0139E36f / floorf(+1.8133E35f))))))) {
  if (comp <= acosf((-1.5215E-30f - atanf((-1.6059E-43f / var_5 * +1.3769E-42f + (var_6 + var_7 + +1.5183E-37f)))))) {
    for (int i=0; i < var_1; ++i) {
      comp += powf((var_8 / (var_9 * -0.0f)), +1.6931E-41f);
for (int i=0; i < var_2; ++i) {
  float tmp_1 = -1.9088E-41f;
var_10[i] = +1.5159E-42f;
comp += var_10[i] + tmp_1 - (+1.5150E-43f * -1.1537E36f + (var_11 / var_12 / -0.0f));
comp = (var_13 / coshf(-1.4654E-43f + +1.1435E-42f));
}
for (int i=0; i < var_3; ++i) {
  comp = (var_14 / (var_15 - coshf(acosf(sinhf((-1.3010E-44f / var_16))))));
}
}
}
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  int tmp_4 = atoi(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float* tmp_11 = initPointer( atof(argv[11]) );
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17);
  hipDeviceSynchronize();

  return 0;
}
