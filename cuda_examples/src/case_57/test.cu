
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, float var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9) {
if (comp > -0.0f * sinhf((var_1 - (+1.6317E-42f + +0.0f / fmodf(+1.8178E-42f * fabsf(-1.2253E36f), +1.5575E-35f * var_2))))) {
  if (comp >= (var_3 + var_4 / var_5)) {
    float tmp_1 = +0.0f;
comp += tmp_1 / -1.8984E-35f + (var_6 / atanf(var_7 - var_8 / var_9));
}
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10);
  hipDeviceSynchronize();

  return 0;
}
