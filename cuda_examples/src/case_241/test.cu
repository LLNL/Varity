
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, int var_1,int var_2,int var_3,int var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float* var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20,float* var_21,float* var_22) {
for (int i=0; i < var_1; ++i) {
  for (int i=0; i < var_2; ++i) {
    if (comp < var_5 + var_6 / sinhf((-1.9525E-42f / -1.9125E-43f + -1.7595E-14f))) {
      float tmp_1 = (var_7 / sinhf(-0.0f / (var_8 - ldexpf((+1.0336E-41f / -1.7989E-35f + sqrtf(sqrtf(var_9 + var_10 - var_11 * +1.1901E-15f))), 2))));
comp = tmp_1 + powf(+1.3671E-2f / (+1.4016E35f / +0.0f), +1.1971E-6f);
for (int i=0; i < var_3; ++i) {
  var_12[i] = expf(+0.0f / log10f(+1.2910E-43f));
comp += var_12[i] + (+1.4961E15f + var_13 + var_14);
}
if (comp < sinhf(-0.0f * (+1.7127E-37f / +1.6857E-37f))) {
  comp = +1.0177E35f * ceilf(atanf((var_15 + var_16 - fmodf(+1.9858E-19f * +1.3827E34f, (var_17 * (-0.0f / var_18))))));
comp += fabsf(var_19 + -1.2918E36f - +1.3146E-36f * (+0.0f * -1.1836E-42f));
comp = (-1.6744E-41f * fabsf(+1.4058E34f * (-1.0385E-41f * (+1.7333E36f - (var_20 / +0.0f)))));
}
for (int i=0; i < var_4; ++i) {
  var_21[i] = (-1.5486E7f / (+1.6543E-43f - -1.6441E-36f));
var_22[i] = +1.9275E35f;
comp += var_22[i] / var_21[i] * -0.0f / +1.5934E2f;
comp = -1.9194E35f + +1.0712E34f;
}
}
}
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  int tmp_4 = atoi(argv[4]);
  int tmp_5 = atoi(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float* tmp_13 = initPointer( atof(argv[13]) );
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float* tmp_22 = initPointer( atof(argv[22]) );
  float* tmp_23 = initPointer( atof(argv[23]) );

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23);
  hipDeviceSynchronize();

  return 0;
}
