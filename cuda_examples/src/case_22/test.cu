
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, float var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8) {
if (comp > (var_1 * coshf(+1.4495E-35f))) {
  float tmp_1 = -1.5411E-43f;
comp += tmp_1 * floorf((-0.0f * (var_2 + (var_3 - +1.9494E22f))));
comp += +1.9190E-12f / sqrtf(+1.1884E-37f);
if (comp <= (+1.5333E6f + asinf(acosf(var_4 * (-0.0f * var_5))))) {
  float tmp_2 = +1.4419E35f;
float tmp_3 = +1.9923E-42f;
comp = tmp_3 * tmp_2 / (var_6 / cosf(-1.7798E-36f + var_7 / (var_8 + (+1.5801E34f * +0.0f))));
}
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9);
  hipDeviceSynchronize();

  return 0;
}
