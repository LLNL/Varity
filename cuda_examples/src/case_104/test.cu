
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, float var_1,float var_2,float var_3,float var_4) {
if (comp > var_1 + log10f(-1.4101E36f)) {
  float tmp_1 = (-1.7403E-43f * -1.3132E-41f + sinhf(+1.9582E19f - var_2 * floorf((-0.0f + (var_3 * (+1.7918E-37f + +1.0935E-11f))))));
comp = tmp_1 + (var_4 / -1.1573E14f);
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5);
  hipDeviceSynchronize();

  return 0;
}
