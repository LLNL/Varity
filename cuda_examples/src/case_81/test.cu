
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, int var_1,int var_2,float var_3,float var_4,int var_5,float var_6,float var_7,float var_8,float var_9,float var_10) {
if (comp == (+1.1633E-35f / ceilf(var_3 / +1.3331E35f * var_4))) {
  for (int i=0; i < var_1; ++i) {
    for (int i=0; i < var_2; ++i) {
      comp = floorf(-1.1702E36f);
for (int i=0; i < var_5; ++i) {
  comp = logf((+0.0f * (var_6 / (var_7 - -0.0f / -1.0583E13f))));
}
if (comp == (-1.2608E34f - var_8)) {
  float tmp_1 = +1.3991E35f;
float tmp_2 = +0.0f / (+1.4080E-43f + var_9);
comp = tmp_2 / tmp_1 / sqrtf((-1.7990E36f / var_10));
}
}
}
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  int tmp_6 = atoi(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11);
  hipDeviceSynchronize();

  return 0;
}
