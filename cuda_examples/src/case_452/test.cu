
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, float var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10) {
if (comp < +1.1187E-37f * (-0.0f * (var_1 / (+0.0f / +0.0f)))) {
  if (comp > (+1.1271E-1f + var_2 * (-1.4708E-9f * +0.0f))) {
    comp = atanf((-1.0526E-41f * (var_3 / +1.9308E-5f / (var_4 + -1.7414E-41f))));
if (comp <= fabsf(var_5 + acosf(+1.0402E-3f))) {
  comp += fabsf(var_6 / (var_7 + (+1.4705E36f / sinhf((var_8 * (var_9 + -0.0f))))));
comp = (var_10 * -1.6885E-44f / +1.8245E34f);
}
}
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11);
  hipDeviceSynchronize();

  return 0;
}
