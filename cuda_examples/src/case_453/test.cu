
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4,float var_5,float var_6) {
if (comp > (var_2 / +0.0f)) {
  for (int i=0; i < var_1; ++i) {
    comp += (-1.7167E-41f / (+0.0f - (var_3 - -1.5503E-35f)));
if (comp > (+1.2504E36f * coshf(+1.2904E-42f * var_4))) {
  float tmp_1 = tanhf((-1.5664E-8f / var_5 / (-1.3954E35f / -1.1985E-35f)));
float tmp_2 = -1.7700E35f;
float tmp_3 = -1.6487E-13f;
comp = tmp_3 - tmp_2 + tmp_1 + (-1.5199E17f * (var_6 / +1.2018E-35f));
}
}
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7);
  hipDeviceSynchronize();

  return 0;
}
