
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4,float* var_5,int var_6,int var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20,float var_21,float var_22,float var_23,float var_24,float var_25,float var_26,float var_27,float var_28,float var_29) {
if (comp == (+0.0f - -1.2479E35f / (-1.8671E36f / coshf((+0.0f * (-0.0f + var_2 + (var_3 + var_4))))))) {
  if (comp < -1.5390E-42f / (-0.0f + floorf((-1.1163E17f - -1.4826E-36f)))) {
    for (int i=0; i < var_1; ++i) {
      float tmp_1 = -1.7034E-35f;
var_5[i] = sinf(+1.0246E35f);
comp = var_5[i] * tmp_1 - sqrtf(var_8 / -0.0f);
for (int i=0; i < var_6; ++i) {
  comp = sqrtf((-1.2861E-36f * sinf((+1.9299E17f - var_9 / +1.9421E-37f * var_10))));
comp = (var_11 * -1.6041E-25f - (var_12 * +0.0f - var_13));
}
for (int i=0; i < var_7; ++i) {
  comp = +1.8307E-18f * (-1.1635E-8f / -0.0f);
comp = (-0.0f * (var_14 * (var_15 - +0.0f - (var_16 / var_17))));
comp = sinhf((var_18 / acosf((var_19 + logf((var_20 - -0.0f))))));
}
if (comp > var_21 - expf(-1.0257E-36f)) {
  comp += (var_22 - var_23 - (-0.0f + var_24));
float tmp_2 = (+1.3188E35f / var_25 + sqrtf(var_26 * var_27 / -1.1334E-44f));
comp = tmp_2 - var_28 / (-1.7613E36f / -0.0f * var_29);
}
}
}
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float* tmp_6 = initPointer( atof(argv[6]) );
  int tmp_7 = atoi(argv[7]);
  int tmp_8 = atoi(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);
  float tmp_23 = atof(argv[23]);
  float tmp_24 = atof(argv[24]);
  float tmp_25 = atof(argv[25]);
  float tmp_26 = atof(argv[26]);
  float tmp_27 = atof(argv[27]);
  float tmp_28 = atof(argv[28]);
  float tmp_29 = atof(argv[29]);
  float tmp_30 = atof(argv[30]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23,tmp_24,tmp_25,tmp_26,tmp_27,tmp_28,tmp_29,tmp_30);
  hipDeviceSynchronize();

  return 0;
}
