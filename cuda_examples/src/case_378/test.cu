
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, float var_1,float var_2,float var_3,float var_4,float var_5) {
if (comp <= +1.5806E-35f * var_1) {
  comp += var_2 * (-1.8370E-5f - (var_3 - var_4 * +1.7035E8f));
comp += -1.0182E-36f / (-1.5508E-5f / cosf((-1.3974E-37f + fabsf((+1.2100E34f + (+1.6483E35f + (var_5 + +1.0791E-41f)))))));
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6);
  hipDeviceSynchronize();

  return 0;
}
