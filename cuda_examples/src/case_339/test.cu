
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, int var_1,int var_2,float var_3,float var_4,float var_5) {
for (int i=0; i < var_1; ++i) {
  if (comp < (+0.0f + var_3 * var_4)) {
    for (int i=0; i < var_2; ++i) {
      comp = (+1.0414E36f + (-1.9003E35f + (var_5 - -1.6587E-35f)));
}
}
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6);
  hipDeviceSynchronize();

  return 0;
}
