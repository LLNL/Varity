
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, int var_1,float var_2,int var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12) {
for (int i=0; i < var_1; ++i) {
  if (comp >= atan2f(acosf(+0.0f - (-1.6874E-42f / (-1.8060E36f / (+1.6270E-44f + var_2)))), (+1.6197E34f - +1.4289E34f * +0.0f / -0.0f))) {
    comp = (+1.8114E-37f / ceilf(var_4 + var_5 / +1.5925E34f - var_6));
if (comp == coshf(-1.7999E-35f)) {
  comp = var_7 - var_8 - asinf(ldexpf(-1.3128E35f, 2));
}
for (int i=0; i < var_3; ++i) {
  comp = ldexpf((var_9 - var_10 - var_11), 2);
comp = +1.6771E-35f * var_12 / +1.6151E-19f;
}
}
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  int tmp_4 = atoi(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13);
  hipDeviceSynchronize();

  return 0;
}
