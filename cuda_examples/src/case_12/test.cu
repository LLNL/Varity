
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, float var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17) {
if (comp == +1.0347E-36f - atan2f(sinf(+1.4337E35f), (+1.3940E36f + var_1 + (var_2 * -1.5186E36f + var_3)))) {
  if (comp > (var_4 * sqrtf(+1.7793E34f))) {
    if (comp >= -1.0748E-42f * (-1.4484E-41f + (-1.9524E-43f + (+1.6624E-25f * -0.0f)))) {
      if (comp >= (var_5 - floorf(var_6 + -1.5508E28f * (-1.8030E-41f / +1.6465E-43f)))) {
        comp = (-1.9321E-16f * var_7);
if (comp >= sqrtf(+1.6382E-35f)) {
  float tmp_1 = -1.5351E-42f;
float tmp_2 = var_8 * (var_9 - -1.0149E36f / var_10 * +1.6542E-43f * -1.7481E21f);
comp = tmp_2 + tmp_1 + (+1.6733E34f * (+1.7556E36f * var_11));
comp += (var_12 / cosf((var_13 - var_14 + ceilf((var_15 + var_16 * var_17)))));
}
}
}
}
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18);
  hipDeviceSynchronize();

  return 0;
}
