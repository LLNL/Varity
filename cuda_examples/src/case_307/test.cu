
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, int var_1,float var_2,float var_3,int var_4,int var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20,float* var_21,float var_22,float var_23,float var_24) {
for (int i=0; i < var_1; ++i) {
  if (comp == -1.9905E-35f + var_2 / var_3) {
    comp = ceilf(+1.2790E-35f + (+0.0f * var_6 - var_7 + var_8));
comp = +0.0f * fmodf(-1.9743E35f, (var_9 * var_10 * +1.7138E34f + +1.3439E-44f));
comp += fmodf(-1.6418E-35f + (+0.0f + -0.0f * (var_11 - -1.2465E-37f + -1.6758E35f)), var_12 - (+1.2379E-35f / powf(atan2f(-1.5887E36f, atan2f((var_13 * (var_14 + var_15 + -1.7684E-43f + +1.7567E-35f)), (+1.0763E-36f / -0.0f * -1.5398E36f))), (var_16 * var_17))));
for (int i=0; i < var_4; ++i) {
  comp += (+0.0f - var_18 * +1.6596E35f * (var_19 * -0.0f / +1.2458E-7f));
float tmp_1 = +0.0f;
comp += tmp_1 * sinhf(-0.0f * -0.0f - var_20);
}
for (int i=0; i < var_5; ++i) {
  var_21[i] = (var_22 + +1.2177E-42f + atan2f(-1.0980E-41f, -1.4250E35f));
comp += var_21[i] + var_23 * var_24 + +1.7340E-37f + floorf((-0.0f / (-1.8883E-37f * +1.6865E34f)));
}
}
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  int tmp_5 = atoi(argv[5]);
  int tmp_6 = atoi(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float* tmp_22 = initPointer( atof(argv[22]) );
  float tmp_23 = atof(argv[23]);
  float tmp_24 = atof(argv[24]);
  float tmp_25 = atof(argv[25]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23,tmp_24,tmp_25);
  hipDeviceSynchronize();

  return 0;
}
