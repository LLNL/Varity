
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4,float var_5,float var_6) {
for (int i=0; i < var_1; ++i) {
  if (comp <= -1.1118E-44f / (-1.5740E-44f / (var_2 * -1.2929E-37f - var_3 / +1.6387E-37f))) {
    float tmp_1 = (+1.4596E34f - (var_4 * ldexpf(var_5 + +1.4768E-37f, 2)));
comp = tmp_1 + expf(var_6 / -1.0160E19f + sinhf(+1.2524E36f));
}
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7);
  hipDeviceSynchronize();

  return 0;
}
