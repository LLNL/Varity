
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, float var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7) {
if (comp == -1.3619E-37f + acosf(sinhf(acosf(atanf((-0.0f + (var_1 / (+1.1494E-43f - var_2 * (var_3 * -0.0f))))))))) {
  comp = ldexpf((var_4 - -1.2794E-44f), 2);
comp += (var_5 * var_6 / +1.4627E14f * (var_7 / (+0.0f - -1.6540E34f)));
comp = (+1.7567E-42f + +1.0899E-44f + -1.6225E6f);
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8);
  hipDeviceSynchronize();

  return 0;
}
