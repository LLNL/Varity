
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, float var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15) {
if (comp <= (+0.0f - var_1)) {
  comp += (var_2 / sinhf((+1.7583E34f - cosf(ldexpf((-1.3731E35f / -1.9426E-35f - +1.2726E34f), 2)))));
comp = (-1.6435E-41f / -1.9180E-44f * -1.1709E-43f + +1.6432E-11f);
float tmp_1 = var_3 / (var_4 * (-1.9631E-41f * +1.2510E4f + -1.4633E35f / -0.0f));
comp += tmp_1 - -1.1638E-36f / (var_5 / (+0.0f / (var_6 - +0.0f)));
if (comp > (-1.8237E-43f - (var_7 / (var_8 + var_9)))) {
  comp = atan2f(var_10 / var_11 - tanhf(expf(+1.0671E28f)), +1.3638E-35f);
float tmp_2 = var_12 - sinf(asinf((var_13 - floorf(var_14 + var_15))));
float tmp_3 = +1.6033E-9f;
comp += tmp_3 - tmp_2 / ceilf(acosf(-0.0f));
}
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16);
  hipDeviceSynchronize();

  return 0;
}
