
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, float var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20,float var_21,float var_22) {
comp = var_1 + (var_2 * +1.1557E-41f * (var_3 + var_4));
float tmp_1 = (var_5 * ldexpf(var_6 + var_7 * -1.7301E-20f / sinf(+0.0f * (-1.8301E-43f - (-1.4620E-37f - var_8 + var_9))), 2));
comp += tmp_1 - var_10 - (var_11 / -1.5235E2f + +1.9869E34f * -0.0f);
comp = (var_12 * (+1.7208E-41f * var_13));
if (comp >= +1.1334E35f / atanf(var_14 + -0.0f)) {
  float tmp_2 = (var_15 * -1.1243E-24f - +1.0263E35f + +1.6528E34f + -1.6945E34f);
comp += tmp_2 * var_16 - var_17;
comp += -1.9974E-43f + var_18;
comp += (var_19 + -1.0657E-37f + (var_20 * cosf(var_21 - var_22)));
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);
  float tmp_23 = atof(argv[23]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23);
  hipDeviceSynchronize();

  return 0;
}
