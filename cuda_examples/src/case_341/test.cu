
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, int var_1,int var_2,float var_3,float var_4,float var_5,float var_6,int var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float* var_20,float var_21,float var_22,float var_23) {
for (int i=0; i < var_1; ++i) {
  for (int i=0; i < var_2; ++i) {
    if (comp > (var_3 + (-1.0098E-42f * var_4 / ldexpf((-1.0995E-35f * log10f((var_5 / -0.0f / (var_6 - (-1.2510E-29f / +0.0f))))), 2)))) {
      comp += sqrtf((+1.3200E-27f + var_8 / -0.0f));
comp += sinhf(var_9 + expf((-1.5029E-36f + (+1.1308E34f * (+1.6593E35f * (-1.3221E34f / logf((var_10 / var_11))))))));
if (comp > (var_12 * (var_13 - logf((-1.0601E-42f / -1.8378E-44f / var_14 - -0.0f - +1.2988E-24f))))) {
  comp += var_15 - -1.1625E-36f * +1.0075E6f * (var_16 / (-1.7412E21f * +1.7537E36f));
comp += (-1.9807E-44f + var_17 / (var_18 + powf(-1.3309E19f, (+1.0554E-41f - (var_19 + -1.4991E-44f)))));
}
for (int i=0; i < var_7; ++i) {
  var_20[i] = -1.2342E35f;
comp += var_20[i] - atanf((var_21 / ceilf(sqrtf(floorf(sqrtf(var_22 / var_23))))));
}
}
}
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  int tmp_8 = atoi(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float* tmp_21 = initPointer( atof(argv[21]) );
  float tmp_22 = atof(argv[22]);
  float tmp_23 = atof(argv[23]);
  float tmp_24 = atof(argv[24]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23,tmp_24);
  hipDeviceSynchronize();

  return 0;
}
