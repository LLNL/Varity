
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, float var_1,float var_2) {
float tmp_1 = +0.0f;
comp += tmp_1 - (var_1 - (var_2 - +1.4829E-37f - -1.4228E-44f));
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  float tmp_3 = atof(argv[3]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3);
  hipDeviceSynchronize();

  return 0;
}
