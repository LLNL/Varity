
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, int var_1,float* var_2,float* var_3,int var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20,float var_21,float var_22,float var_23,float var_24,float var_25,float var_26,float var_27,float var_28) {
for (int i=0; i < var_1; ++i) {
  float tmp_1 = -1.0702E-35f;
var_2[i] = +1.7395E-42f;
var_3[i] = +1.7427E-35f / -1.9251E-19f - (-1.0082E-36f - (var_5 - var_6 / var_7));
comp += var_3[i] / var_2[i] + tmp_1 * (-1.0329E-35f + (+1.4465E-42f * (+1.0312E34f / (var_8 + var_9))));
if (comp == -1.7099E-41f / var_10 / asinf(var_11 + (-1.7817E-43f - var_12 + var_13))) {
  comp = var_14 * (+1.4463E-43f / (var_15 * (-0.0f / +1.8534E-20f - -1.7533E-25f)));
comp += var_16 + -0.0f / -1.8414E21f / +1.3469E35f;
comp = (+1.8135E18f / var_17);
}
if (comp == var_18 * log10f(var_19 * logf((var_20 + (-1.7673E-35f * (var_21 * var_22)))))) {
  comp += (+1.5038E-35f / (var_23 + var_24));
comp += +1.9786E-43f + var_25 - +0.0f;
}
for (int i=0; i < var_4; ++i) {
  comp = var_26 / var_27;
comp += (var_28 + logf(+1.1288E-37f));
}
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float* tmp_3 = initPointer( atof(argv[3]) );
  float* tmp_4 = initPointer( atof(argv[4]) );
  int tmp_5 = atoi(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);
  float tmp_23 = atof(argv[23]);
  float tmp_24 = atof(argv[24]);
  float tmp_25 = atof(argv[25]);
  float tmp_26 = atof(argv[26]);
  float tmp_27 = atof(argv[27]);
  float tmp_28 = atof(argv[28]);
  float tmp_29 = atof(argv[29]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23,tmp_24,tmp_25,tmp_26,tmp_27,tmp_28,tmp_29);
  hipDeviceSynchronize();

  return 0;
}
