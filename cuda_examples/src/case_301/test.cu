
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, float var_1,float var_2,float var_3) {
if (comp >= (var_1 / +1.6871E36f + (var_2 * var_3 * +1.1162E-35f))) {
  comp += atanf(-1.5007E35f * -1.7422E-5f - -1.2423E36f);
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4);
  hipDeviceSynchronize();

  return 0;
}
