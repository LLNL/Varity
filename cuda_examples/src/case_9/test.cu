
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4,int var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15) {
for (int i=0; i < var_1; ++i) {
  if (comp >= (var_2 * -1.7511E35f - (var_3 - var_4))) {
    comp += (var_6 - acosf(+1.9495E35f));
if (comp < (var_7 + +1.2086E-9f)) {
  float tmp_1 = +0.0f;
comp = tmp_1 / var_8 / (-1.3261E-35f * logf((var_9 * (-1.3095E-26f + var_10))));
comp = (var_11 * +1.7135E-43f);
}
for (int i=0; i < var_5; ++i) {
  comp = asinf((-1.8138E10f * var_12 * (var_13 * atan2f(var_14 - var_15, +0.0f + -1.3810E-36f * -1.0811E34f + +1.9882E-37f))));
}
}
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  int tmp_6 = atoi(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16);
  hipDeviceSynchronize();

  return 0;
}
