
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, float var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10) {
if (comp <= (var_1 * powf((var_2 / var_3 - -1.2661E36f * tanhf((-1.2654E-35f / (-1.6959E26f * +1.3592E35f)))), acosf((-1.9116E-36f + +0.0f))))) {
  if (comp == (var_4 + var_5 * var_6 - var_7)) {
    if (comp <= floorf((+1.9235E-37f + +1.2517E36f - (var_8 / (var_9 + -1.0885E15f))))) {
      float tmp_1 = -1.1715E-36f;
comp = tmp_1 / (-1.7614E36f / +1.0835E-22f * (-0.0f * (+1.5859E34f - var_10)));
}
}
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11);
  hipDeviceSynchronize();

  return 0;
}
