
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11) {
if (comp < expf((var_2 - (+1.5264E-22f / tanhf(sinhf(+1.7187E34f - var_3 + -1.3384E-41f)))))) {
  for (int i=0; i < var_1; ++i) {
    float tmp_1 = -1.5386E-37f / acosf(-1.0170E29f * floorf(-0.0f * sinf(var_4 - -0.0f)));
comp += tmp_1 - acosf(-1.3608E-37f);
float tmp_2 = -1.7865E-42f;
comp = tmp_2 / sqrtf((-1.1519E-36f / atanf((var_5 + (-1.7364E-37f / (+1.2048E-36f - -1.3147E36f))))));
if (comp < var_6 * (+1.4910E-1f - sinhf(var_7 * -1.1515E-36f / +1.0154E36f))) {
  float tmp_3 = var_8 - (+1.6522E35f - var_9 + -1.2971E28f - var_10 * -1.6359E36f);
comp += tmp_3 / (-0.0f * var_11);
}
}
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12);
  hipDeviceSynchronize();

  return 0;
}
