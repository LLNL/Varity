
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, int var_1,int var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15) {
for (int i=0; i < var_1; ++i) {
  if (comp == acosf(+1.7250E35f)) {
    comp += (var_3 / var_4 - var_5 / var_6);
if (comp > (+1.2638E36f - ldexpf(-1.6108E34f - (var_7 - +1.7065E-19f), 2))) {
  comp = sqrtf(+1.4345E35f + +0.0f);
comp += (var_8 / -1.7229E34f / (-1.5260E35f - var_9 + -1.7803E3f));
}
for (int i=0; i < var_2; ++i) {
  comp = -1.3042E-44f - (var_10 * var_11);
comp = (var_12 * (var_13 - var_14 / +1.3264E8f));
}
if (comp > -1.8163E-29f - +1.0793E-16f) {
  float tmp_1 = -0.0f - var_15;
comp = tmp_1 - -1.4916E3f / +0.0f / +1.6909E10f;
}
}
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16);
  hipDeviceSynchronize();

  return 0;
}
