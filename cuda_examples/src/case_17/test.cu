
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, float var_1,float var_2,float var_3,float var_4,int var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17) {
if (comp < var_1 * -0.0f) {
  if (comp == var_2 * var_3 + -0.0f + (var_4 / (-1.6899E-43f * -1.1486E-42f))) {
    float tmp_1 = +1.5608E-43f;
comp += tmp_1 + (var_6 * +0.0f - +1.0744E-21f - +1.2833E-37f / (-1.3369E-35f * -1.4943E-43f));
comp = var_7 * var_8 * var_9 / (+1.3722E-37f / +0.0f);
for (int i=0; i < var_5; ++i) {
  comp = asinf(-1.9620E36f);
}
if (comp < -1.2650E35f + (var_10 - -1.3543E36f / var_11 + -1.7585E-43f)) {
  float tmp_2 = +0.0f - var_12 - -1.4642E-36f;
float tmp_3 = +1.3939E-42f;
comp += tmp_3 / tmp_2 - var_13 + sqrtf((-1.1327E-42f * atan2f(var_14 * coshf(-1.2476E-42f + -1.7998E35f / (var_15 - (-1.9949E-7f / (-1.2379E-36f / +1.6939E34f)))), -1.9986E-36f - var_16 * var_17 - +1.2399E34f + (-1.0323E13f - +1.9440E35f))));
}
}
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  int tmp_6 = atoi(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18);
  hipDeviceSynchronize();

  return 0;
}
