
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17) {
for (int i=0; i < var_1; ++i) {
  if (comp < var_2 + atanf((-1.9475E-11f + (var_3 - (var_4 + var_5 - -1.8930E34f))))) {
    float tmp_1 = -1.7769E-28f;
float tmp_2 = (var_6 + (var_7 * +1.8717E35f));
comp += tmp_2 * tmp_1 / +0.0f * -1.8600E-35f + tanhf((var_8 * var_9));
comp = (var_10 / (var_11 + -1.5789E16f));
if (comp == (-1.6910E-14f * +0.0f * -1.3167E-35f)) {
  float tmp_3 = (+1.6249E24f - (var_12 * atan2f(+1.3794E-42f, (var_13 * (-0.0f + (var_14 - var_15 * var_16))))));
comp = tmp_3 * (var_17 / (-1.0536E-27f - -1.3789E-29f + +1.2507E36f));
}
}
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18);
  hipDeviceSynchronize();

  return 0;
}
