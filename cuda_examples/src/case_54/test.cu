
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, float var_1) {
if (comp > cosf((-1.2114E34f * +1.6991E-28f * (+1.6129E-37f - -1.0721E-44f)))) {
  comp = (-0.0f + -0.0f / var_1 + (-0.0f - -1.2560E12f));
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);

  compute<<<1,1>>>(tmp_1,tmp_2);
  hipDeviceSynchronize();

  return 0;
}
