
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, float var_1,float var_2,float var_3,int var_4,int var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20,float var_21,float var_22) {
if (comp < var_1 * (-0.0f - -1.0488E-41f)) {
  if (comp >= (+1.6809E-44f / atanf(+0.0f + (var_2 + var_3 + -1.9413E35f)))) {
    float tmp_1 = var_6 / var_7 * logf(floorf(-1.8625E-14f));
float tmp_2 = -1.4014E-35f;
comp += tmp_2 / tmp_1 + (+1.1014E14f - var_8 - -1.0898E-43f);
for (int i=0; i < var_4; ++i) {
  comp = var_9 + powf((+1.8927E36f * (var_10 * coshf(var_11 / (var_12 - -1.2219E-44f)))), var_13 + (+1.1021E13f / expf((-1.4460E-35f + -0.0f))));
}
for (int i=0; i < var_5; ++i) {
  comp += (-1.4263E-41f / +1.4689E-37f * -1.7224E26f);
comp = var_14 * (+0.0f / atan2f(+1.8768E-42f, sinf(acosf(+0.0f))));
}
if (comp <= var_15 / var_16 + (-1.8000E-44f + +1.3862E-35f)) {
  float tmp_3 = +0.0f;
comp += tmp_3 * var_17 / (-1.1685E-1f * -1.1878E35f + +1.0889E34f);
float tmp_4 = -1.3936E36f;
comp = tmp_4 * fmodf(var_18 / (-1.4626E-11f * log10f(-0.0f)), var_19 - (var_20 - -1.9467E-26f * (var_21 - (var_22 * +1.4935E35f))));
}
}
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  int tmp_5 = atoi(argv[5]);
  int tmp_6 = atoi(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);
  float tmp_23 = atof(argv[23]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23);
  hipDeviceSynchronize();

  return 0;
}
