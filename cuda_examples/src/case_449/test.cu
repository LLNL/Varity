
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, float var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7) {
if (comp <= (+1.6716E-44f / -1.5155E23f / ceilf(sinf(fmodf(+1.2273E35f, var_1 + (var_2 * (-1.3112E35f / var_3))))))) {
  comp = (+1.2672E-36f + (+1.3490E-37f / (var_4 / (var_5 + -1.3174E-37f - +1.5944E-44f))));
comp = (-1.0648E36f + ldexpf(-1.4353E-37f, 2));
comp += +1.1875E36f * var_6 / coshf(+1.0999E-36f / (+1.1275E34f / var_7));
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8);
  hipDeviceSynchronize();

  return 0;
}
