
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, float var_1,int var_2,float var_3,float var_4,float var_5,float var_6,float* var_7,float* var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20) {
if (comp == ceilf((-0.0f - (-1.8873E35f - var_1)))) {
  float tmp_1 = sqrtf(+1.5753E10f);
comp = tmp_1 + var_3 + var_4 + expf((+0.0f / (+1.5338E-44f - var_5 - (var_6 * (+1.6869E34f * -1.4145E-41f)))));
for (int i=0; i < var_2; ++i) {
  var_7[i] = +0.0f;
var_8[i] = -1.9246E-37f;
comp += var_8[i] / var_7[i] * logf(var_9 + sinhf(var_10 + (var_11 * var_12 + -1.5616E34f - var_13)));
}
if (comp >= -1.9703E-36f + (-1.6207E-42f - (var_14 / var_15 / +0.0f))) {
  comp = -1.4078E-13f / -0.0f * (var_16 - (var_17 / var_18));
comp += sqrtf(tanhf(+0.0f + (var_19 - ldexpf(var_20 / +1.6254E-25f, 2))));
}
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  int tmp_3 = atoi(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float* tmp_8 = initPointer( atof(argv[8]) );
  float* tmp_9 = initPointer( atof(argv[9]) );
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21);
  hipDeviceSynchronize();

  return 0;
}
