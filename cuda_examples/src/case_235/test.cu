
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, float var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15) {
if (comp == (+1.9838E-35f / tanhf(+1.4160E11f))) {
  comp = var_1 * (var_2 - var_3);
comp = -1.4350E-29f * cosf((var_4 / (var_5 - -0.0f)));
comp = (var_6 + var_7);
if (comp >= (+1.5993E-19f * atan2f(-1.3948E-42f * (var_8 - (-0.0f + +1.4240E6f + +1.9767E-35f / -1.8233E35f)), (var_9 - fmodf(var_10 * +1.8475E-37f * (var_11 * var_12 * tanhf(-1.7305E36f)), -1.6347E35f))))) {
  comp = (+1.7185E-35f / (-1.6289E-17f * +1.3788E-41f - var_13 + -1.8367E23f));
float tmp_1 = -1.3860E-37f;
comp += tmp_1 + (-1.8416E34f / log10f(var_14 + var_15));
comp = ceilf(cosf(-0.0f));
}
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16);
  hipDeviceSynchronize();

  return 0;
}
