
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, float var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13) {
if (comp > (-1.4090E1f - (var_1 * -1.9889E-9f - (var_2 - var_3)))) {
  if (comp <= sinhf(-1.2483E-43f / var_4 * (-1.1176E-28f - (+1.7369E35f * -1.1525E-37f)))) {
    if (comp > atan2f(+1.5106E-42f, atan2f((var_5 / -1.1533E-43f + (var_6 + atanf(tanhf(-1.5643E-17f)))), var_7 + (+1.3014E10f - var_8)))) {
      float tmp_1 = +0.0f;
comp = tmp_1 + (+1.7689E36f / (+0.0f * -1.8398E36f * var_9 - +1.1815E-41f));
if (comp == +1.9968E-37f / var_10) {
  comp += var_11 - var_12 * var_13;
}
}
}
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14);
  hipDeviceSynchronize();

  return 0;
}
