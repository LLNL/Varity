
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, float var_1,float var_2,float var_3,float var_4,int var_5,int var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float* var_20) {
if (comp <= var_1 - (var_2 - (var_3 * var_4))) {
  comp = (-1.4113E-35f + -1.7121E35f / -1.5316E-36f + (var_7 * var_8 + var_9));
float tmp_1 = +1.1032E34f;
comp = tmp_1 - var_10 - sqrtf(-1.4123E29f - (var_11 * floorf((var_12 / +0.0f - -0.0f))));
comp += (var_13 - (+1.5771E5f - (var_14 - -1.0910E-43f * -0.0f)));
for (int i=0; i < var_5; ++i) {
  comp += var_15 / (+1.3782E-36f / var_16 * +0.0f - var_17);
comp += (var_18 / (+1.8357E-44f - var_19));
}
for (int i=0; i < var_6; ++i) {
  var_20[i] = +0.0f;
comp = var_20[i] - -1.8295E-41f * (-0.0f - +0.0f + -1.0260E5f);
}
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  int tmp_6 = atoi(argv[6]);
  int tmp_7 = atoi(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float* tmp_21 = initPointer( atof(argv[21]) );

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21);
  hipDeviceSynchronize();

  return 0;
}
