
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, int var_1,int var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float* var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20,float var_21,float* var_22,float var_23,float var_24,float var_25,float var_26,float var_27,float var_28,float var_29,float var_30,float var_31,float var_32,float var_33,float var_34,float var_35) {
if (comp < sqrtf(var_3 / fabsf(var_4 - var_5))) {
  float tmp_1 = +1.0398E-35f;
comp = tmp_1 - (+1.8022E35f * (var_6 + (-1.2891E36f / var_7 * -1.3743E-43f)));
for (int i=0; i < var_1; ++i) {
  var_8[i] = var_9 + (+1.9129E15f / var_10 + var_11 - var_12);
comp += var_8[i] + coshf(ldexpf((-0.0f - atan2f(-1.1634E-35f, (var_13 - var_14 - -0.0f * +1.7666E9f))), 2));
comp += atan2f(+0.0f - tanhf((var_15 * (var_16 + (-1.4854E-23f - var_17)))), var_18 / (+0.0f - -1.5006E-36f / (var_19 + var_20 - var_21)));
}
for (int i=0; i < var_2; ++i) {
  var_22[i] = (var_23 + (-1.6868E-44f - -1.2618E-44f));
comp = var_22[i] - var_24 / tanhf(-1.3568E35f * tanhf(+1.3535E-37f));
comp += var_25 + -0.0f / (-0.0f * (+1.2949E11f + (var_26 * -1.1446E-35f)));
comp += (var_27 * -1.0322E2f);
}
if (comp > var_28 * -0.0f) {
  comp += (var_29 * var_30 - (-1.7216E-37f + var_31 + acosf(-1.6976E-28f)));
comp = cosf((var_32 - var_33 + var_34));
float tmp_2 = -1.5790E36f;
comp = tmp_2 + (-0.0f - var_35);
}
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float* tmp_9 = initPointer( atof(argv[9]) );
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);
  float* tmp_23 = initPointer( atof(argv[23]) );
  float tmp_24 = atof(argv[24]);
  float tmp_25 = atof(argv[25]);
  float tmp_26 = atof(argv[26]);
  float tmp_27 = atof(argv[27]);
  float tmp_28 = atof(argv[28]);
  float tmp_29 = atof(argv[29]);
  float tmp_30 = atof(argv[30]);
  float tmp_31 = atof(argv[31]);
  float tmp_32 = atof(argv[32]);
  float tmp_33 = atof(argv[33]);
  float tmp_34 = atof(argv[34]);
  float tmp_35 = atof(argv[35]);
  float tmp_36 = atof(argv[36]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23,tmp_24,tmp_25,tmp_26,tmp_27,tmp_28,tmp_29,tmp_30,tmp_31,tmp_32,tmp_33,tmp_34,tmp_35,tmp_36);
  hipDeviceSynchronize();

  return 0;
}
