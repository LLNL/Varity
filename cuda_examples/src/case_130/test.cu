
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, float var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14) {
if (comp < var_1 + -1.9098E-13f - (-1.6590E-35f + var_2 / var_3)) {
  comp += var_4 * var_5 + (-0.0f * (-1.9967E-41f / sinhf(-1.1047E-35f - (var_6 * var_7))));
float tmp_1 = -1.9860E-17f;
comp += tmp_1 + var_8 / sqrtf(acosf((-1.9410E-36f - (var_9 / var_10))));
if (comp < log10f(powf(+1.9282E36f * var_11 * (var_12 - (var_13 / +1.0302E-42f)), var_14 - (+0.0f * powf(-0.0f, -1.6385E0f))))) {
  float tmp_2 = atanf(+1.4938E36f / atanf(+1.0958E-35f));
comp = tmp_2 - (+1.6552E36f * +0.0f);
}
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15);
  hipDeviceSynchronize();

  return 0;
}
