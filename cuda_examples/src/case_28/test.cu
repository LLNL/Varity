
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, float var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,int var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20,float* var_21,float* var_22,float var_23,float var_24,float var_25,float var_26,float var_27) {
if (comp >= atan2f(+1.8263E-35f * (-1.4330E-35f + var_1), atanf(-1.7431E-24f - var_2))) {
  if (comp < var_3 * -1.9534E-16f / var_4) {
    if (comp > -1.3650E13f * (+1.4388E10f / (-0.0f + sqrtf(acosf(-1.0829E-37f + (var_5 / var_6 / (var_7 / -1.6908E-44f))))))) {
      float tmp_1 = var_9 + var_10;
float tmp_2 = var_11 * (+1.2320E-44f * var_12);
comp = tmp_2 / tmp_1 / var_13 - var_14;
if (comp > -1.5878E36f * (+1.2811E20f + var_15)) {
  comp += (-1.6012E-35f * atanf(-1.0106E-36f));
float tmp_3 = (-1.8678E-42f - +0.0f / expf(var_16 / atanf((-1.2718E-42f / (var_17 + (+1.1095E-35f * var_18))))));
comp = tmp_3 / -1.0010E-44f - +0.0f;
comp += (var_19 * (-1.4058E-41f * var_20));
}
for (int i=0; i < var_8; ++i) {
  var_21[i] = -1.5802E-37f;
var_22[i] = +1.4171E36f + (+1.0369E-37f - (var_23 - var_24));
comp = var_22[i] - var_21[i] / ceilf(+0.0f - ldexpf((+1.0567E-44f - var_25), 2));
}
if (comp > (var_26 + (+1.6345E-37f + (-1.5608E0f + (+1.0870E35f - var_27))))) {
  comp += (-1.2127E-41f * -1.5377E15f);
}
}
}
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  int tmp_9 = atoi(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float* tmp_22 = initPointer( atof(argv[22]) );
  float* tmp_23 = initPointer( atof(argv[23]) );
  float tmp_24 = atof(argv[24]);
  float tmp_25 = atof(argv[25]);
  float tmp_26 = atof(argv[26]);
  float tmp_27 = atof(argv[27]);
  float tmp_28 = atof(argv[28]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23,tmp_24,tmp_25,tmp_26,tmp_27,tmp_28);
  hipDeviceSynchronize();

  return 0;
}
