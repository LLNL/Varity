
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, int var_1,int var_2,int var_3,int var_4,float var_5,float* var_6,float* var_7,float var_8,float var_9,float var_10) {
for (int i=0; i < var_1; ++i) {
  for (int i=0; i < var_2; ++i) {
    for (int i=0; i < var_3; ++i) {
      if (comp == atanf(+1.0452E-37f)) {
        comp = +1.4594E34f - var_5;
for (int i=0; i < var_4; ++i) {
  var_6[i] = var_8 * +1.1414E34f;
var_7[i] = (-1.0140E29f * var_9 + -1.5541E-43f);
comp += var_7[i] * var_6[i] * var_10 / sinhf(-1.9131E-36f);
comp += atanf(-0.0f);
}
}
}
}
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  int tmp_4 = atoi(argv[4]);
  int tmp_5 = atoi(argv[5]);
  float tmp_6 = atof(argv[6]);
  float* tmp_7 = initPointer( atof(argv[7]) );
  float* tmp_8 = initPointer( atof(argv[8]) );
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11);
  hipDeviceSynchronize();

  return 0;
}
