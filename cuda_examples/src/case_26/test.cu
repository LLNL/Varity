
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, float var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8) {
if (comp <= (var_1 - (-1.8809E-35f - (+1.2500E-42f / var_2 + (-0.0f - var_3))))) {
  if (comp > (var_4 * atanf(fmodf(cosf(-1.1861E-37f), -1.0219E-35f / asinf((var_5 - var_6 + +1.3468E-37f)))))) {
    comp = asinf(atan2f(log10f(var_7 / expf(-1.2621E-35f)), -1.5984E-35f * (-1.0863E-36f / +1.2865E-5f + var_8)));
comp += tanhf(-0.0f);
}
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9);
  hipDeviceSynchronize();

  return 0;
}
