
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, int var_1,int var_2,float var_3,float var_4,float* var_5,float* var_6,float var_7,float var_8,float var_9,float var_10) {
if (comp <= (var_3 - var_4)) {
  for (int i=0; i < var_1; ++i) {
    comp = +1.3954E35f + -0.0f;
for (int i=0; i < var_2; ++i) {
  var_5[i] = +1.7694E35f;
comp += var_5[i] - +1.9550E-43f * +1.6932E-43f - (-1.4210E-35f / (var_7 / -0.0f));
var_6[i] = +1.6805E-35f;
comp += var_6[i] + var_8 / -0.0f * var_9 - var_10;
}
}
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float* tmp_6 = initPointer( atof(argv[6]) );
  float* tmp_7 = initPointer( atof(argv[7]) );
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11);
  hipDeviceSynchronize();

  return 0;
}
