
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17) {
if (comp >= -1.0106E-37f * (var_2 / var_3)) {
  comp = sinf(-0.0f / (+1.3812E-1f * -1.9419E34f * fmodf(+1.8742E35f, var_4 - +1.5845E36f * var_5)));
comp += (-1.0612E-11f + sinf((var_6 + log10f(+1.3991E36f))));
comp += var_7 / (-1.1860E-36f + (-1.7029E5f * var_8 - var_9));
if (comp <= (var_10 / sinf((-1.8483E-36f * (+1.3027E-43f + -1.0000E-35f * var_11 + -1.5438E-37f))))) {
  comp = (-1.8740E-43f - var_12);
}
for (int i=0; i < var_1; ++i) {
  float tmp_1 = (var_13 - -1.0165E26f);
float tmp_2 = +0.0f;
comp = tmp_2 - tmp_1 + var_14 * (+1.0543E-8f / -1.8980E-41f);
comp += -1.6881E-35f / powf((var_15 / -1.0392E8f - var_16 + var_17), acosf(+1.6332E-41f));
}
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18);
  hipDeviceSynchronize();

  return 0;
}
