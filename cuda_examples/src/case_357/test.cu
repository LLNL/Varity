
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, float var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19) {
if (comp == coshf(var_1 * (var_2 - fmodf((+1.2526E34f * (+0.0f * +1.1190E-42f / (var_3 - var_4 - var_5))), -1.5493E-35f)))) {
  if (comp >= -1.8523E36f / (var_6 + (var_7 * (var_8 + +1.5254E35f)))) {
    comp += -1.6531E-35f / (+1.8575E36f + cosf((+1.6701E35f * (var_9 - -1.9499E-41f))));
if (comp == (-1.4244E-12f * (+1.0691E-35f + var_10 * var_11 - atan2f((var_12 - -1.6145E-36f), -1.9264E1f)))) {
  float tmp_1 = (var_13 + (var_14 - (var_15 * var_16 / var_17)));
float tmp_2 = +1.8803E-36f;
comp = tmp_2 / tmp_1 * var_18 / var_19;
}
}
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20);
  hipDeviceSynchronize();

  return 0;
}
