
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, float var_1,float var_2,float var_3) {
if (comp <= (-0.0f - var_1 - +1.9945E-44f / -1.8945E36f)) {
  float tmp_1 = (+1.6933E34f - (-1.3140E20f / +1.2012E-26f));
float tmp_2 = -1.5160E-43f;
comp = tmp_2 / tmp_1 * var_2 * var_3;
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4);
  hipDeviceSynchronize();

  return 0;
}
