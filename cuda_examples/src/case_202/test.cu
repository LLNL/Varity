
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, float var_1,float var_2,float var_3,float var_4,float var_5,int var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20,float var_21,float var_22,float var_23,float var_24,float var_25,float var_26,float var_27,float var_28,float var_29,float var_30,float var_31,float var_32,float var_33,float var_34,float var_35,float var_36,float var_37,float var_38,float var_39,float var_40,float var_41) {
if (comp == var_1 / +0.0f - var_2 - (var_3 * var_4 - var_5)) {
  comp = +1.4329E-43f * (+0.0f / (var_7 / -1.1450E36f));
comp += asinf((+1.5663E-22f / var_8 + logf(ceilf((var_9 + +1.8261E-44f * fmodf((var_10 + sqrtf(+1.6147E-36f)), (+1.2292E35f * (-0.0f - +1.5430E-41f))))))));
comp += (var_11 * sinf((+1.5481E36f / var_12 - atanf(tanhf(-1.8254E-44f)))));
if (comp == (+1.6499E35f * atanf(atanf(var_13 / var_14)))) {
  comp += var_15 - +0.0f / var_16;
float tmp_1 = var_17 * +0.0f - var_18 / tanhf(-1.6246E-36f);
comp += tmp_1 - log10f((+1.3728E-44f / fmodf((var_19 / (var_20 * +0.0f / -1.4205E-35f / +1.7331E-41f)), var_21 + var_22 * (var_23 / +1.4503E34f))));
comp = (+1.9415E34f * (var_24 * +1.0316E-29f * powf(+1.4698E-42f, -1.9273E35f - +1.8795E34f)));
}
if (comp < -1.5516E21f / (+1.6774E35f * (var_25 + -1.9158E36f))) {
  comp += (var_26 * (-1.3676E-37f / (var_27 / var_28)));
float tmp_2 = log10f((var_29 * (+1.6247E-42f / var_30)));
comp += tmp_2 * (-1.5496E-42f / var_31 + (var_32 + var_33 - var_34));
}
for (int i=0; i < var_6; ++i) {
  comp += (+1.8339E-43f + -0.0f * (-1.8858E-41f + (var_35 + expf((var_36 + var_37 / -0.0f / var_38)))));
comp = (-1.1668E-37f + fabsf((+1.9394E34f + var_39 / var_40 + (var_41 * -1.3973E-21f))));
}
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  int tmp_7 = atoi(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);
  float tmp_23 = atof(argv[23]);
  float tmp_24 = atof(argv[24]);
  float tmp_25 = atof(argv[25]);
  float tmp_26 = atof(argv[26]);
  float tmp_27 = atof(argv[27]);
  float tmp_28 = atof(argv[28]);
  float tmp_29 = atof(argv[29]);
  float tmp_30 = atof(argv[30]);
  float tmp_31 = atof(argv[31]);
  float tmp_32 = atof(argv[32]);
  float tmp_33 = atof(argv[33]);
  float tmp_34 = atof(argv[34]);
  float tmp_35 = atof(argv[35]);
  float tmp_36 = atof(argv[36]);
  float tmp_37 = atof(argv[37]);
  float tmp_38 = atof(argv[38]);
  float tmp_39 = atof(argv[39]);
  float tmp_40 = atof(argv[40]);
  float tmp_41 = atof(argv[41]);
  float tmp_42 = atof(argv[42]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23,tmp_24,tmp_25,tmp_26,tmp_27,tmp_28,tmp_29,tmp_30,tmp_31,tmp_32,tmp_33,tmp_34,tmp_35,tmp_36,tmp_37,tmp_38,tmp_39,tmp_40,tmp_41,tmp_42);
  hipDeviceSynchronize();

  return 0;
}
