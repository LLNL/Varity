
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, float var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8) {
if (comp > +1.4454E36f - coshf(var_1 - var_2)) {
  float tmp_1 = -0.0f;
float tmp_2 = -1.2796E36f;
comp = tmp_2 * tmp_1 * var_3 + (+1.6015E-41f + var_4 / +1.2304E23f / powf(atanf(+1.7123E4f), (var_5 * (var_6 * +1.0468E-44f + -1.6785E35f))));
comp = sinf(-0.0f - ceilf((-1.3507E-36f + var_7 - var_8)));
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9);
  hipDeviceSynchronize();

  return 0;
}
