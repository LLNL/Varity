
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, float var_1,float var_2,int var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10) {
if (comp < sqrtf(+1.6099E-43f)) {
  if (comp >= +1.3656E16f * var_1 / var_2 / sinhf(-0.0f + (-1.9300E-42f * +1.3748E-37f))) {
    float tmp_1 = -1.2122E-35f - (+0.0f + +0.0f);
comp = tmp_1 * var_4 - var_5;
float tmp_2 = -1.0392E-43f;
comp += tmp_2 * (var_6 + (var_7 + (var_8 - var_9)));
for (int i=0; i < var_3; ++i) {
  float tmp_3 = +1.9801E34f * log10f((+1.9608E-11f + +0.0f * +0.0f - (var_10 - (-1.1255E-37f * -1.2478E12f))));
float tmp_4 = -1.7650E-36f;
comp = tmp_4 / tmp_3 - -0.0f / +1.4563E-41f;
}
}
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  float tmp_3 = atof(argv[3]);
  int tmp_4 = atoi(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11);
  hipDeviceSynchronize();

  return 0;
}
