
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, float var_1,float var_2,float var_3,float var_4,float var_5,float var_6) {
if (comp == var_1 + var_2 / +1.9016E34f + expf(sinhf(+1.4150E-37f))) {
  if (comp == (+1.9368E-44f / -0.0f + (+1.6143E-37f + var_3))) {
    comp += var_4 - var_5;
if (comp <= (-1.3896E-44f - -1.1397E-20f)) {
  float tmp_1 = -1.5318E-35f;
float tmp_2 = +1.2387E35f;
comp += tmp_2 / tmp_1 * (+0.0f * +1.7532E36f - -0.0f + (-1.6383E36f + -0.0f));
comp += (-1.5410E-36f - (+1.9450E-41f + var_6 * +0.0f));
}
}
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7);
  hipDeviceSynchronize();

  return 0;
}
