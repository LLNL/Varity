
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, float var_1,float var_2,int var_3,float var_4,float var_5,float var_6,float var_7) {
if (comp == cosf(var_1 - +1.8906E35f - (+1.1216E14f / var_2))) {
  comp = (+1.1584E-27f / (var_4 + (-1.9146E35f + log10f((-1.1156E-35f * var_5)))));
comp += -1.8193E-41f + -1.0915E-35f - var_6;
comp += ldexpf(+1.5201E-41f, 2);
for (int i=0; i < var_3; ++i) {
  comp = var_7 * -1.4707E-41f;
}
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  float tmp_3 = atof(argv[3]);
  int tmp_4 = atoi(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8);
  hipDeviceSynchronize();

  return 0;
}
